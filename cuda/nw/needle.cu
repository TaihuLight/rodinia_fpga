#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "needle.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <omp.h>
#include "../../common/timer.h"
#include "../../common/power_gpu.h"

#define LIMIT -999

// includes, kernels
#include "needle_kernel.cu"

TimeStamp start, end;
double totalTime;
int flag = 0;
double power = 0;
double energy = 0;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);


int blosum62[24][24] = {
{ 4, -1, -2, -2,  0, -1, -1,  0, -2, -1, -1, -1, -1, -2, -1,  1,  0, -3, -2,  0, -2, -1,  0, -4},
{-1,  5,  0, -2, -3,  1,  0, -2,  0, -3, -2,  2, -1, -3, -2, -1, -1, -3, -2, -3, -1,  0, -1, -4},
{-2,  0,  6,  1, -3,  0,  0,  0,  1, -3, -3,  0, -2, -3, -2,  1,  0, -4, -2, -3,  3,  0, -1, -4},
{-2, -2,  1,  6, -3,  0,  2, -1, -1, -3, -4, -1, -3, -3, -1,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{ 0, -3, -3, -3,  9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4},
{-1,  1,  0,  0, -3,  5,  2, -2,  0, -3, -2,  1,  0, -3, -1,  0, -1, -2, -1, -2,  0,  3, -1, -4},
{-1,  0,  0,  2, -4,  2,  5, -2,  0, -3, -3,  1, -2, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -2,  0, -1, -3, -2, -2,  6, -2, -4, -4, -2, -3, -3, -2,  0, -2, -2, -3, -3, -1, -2, -1, -4},
{-2,  0,  1, -1, -3,  0,  0, -2,  8, -3, -3, -1, -2, -1, -2, -1, -2, -2,  2, -3,  0,  0, -1, -4},
{-1, -3, -3, -3, -1, -3, -3, -4, -3,  4,  2, -3,  1,  0, -3, -2, -1, -3, -1,  3, -3, -3, -1, -4},
{-1, -2, -3, -4, -1, -2, -3, -4, -3,  2,  4, -2,  2,  0, -3, -2, -1, -2, -1,  1, -4, -3, -1, -4},
{-1,  2,  0, -1, -3,  1,  1, -2, -1, -3, -2,  5, -1, -3, -1,  0, -1, -3, -2, -2,  0,  1, -1, -4},
{-1, -1, -2, -3, -1,  0, -2, -3, -2,  1,  2, -1,  5,  0, -2, -1, -1, -1, -1,  1, -3, -1, -1, -4},
{-2, -3, -3, -3, -2, -3, -3, -3, -1,  0,  0, -3,  0,  6, -4, -2, -2,  1,  3, -1, -3, -3, -1, -4},
{-1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1, -2, -4,  7, -1, -1, -4, -3, -2, -2, -1, -2, -4},
{ 1, -1,  1,  0, -1,  0,  0,  0, -1, -2, -2,  0, -1, -2, -1,  4,  1, -3, -2, -2,  0,  0,  0, -4},
{ 0, -1,  0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1,  1,  5, -2, -2,  0, -1, -1,  0, -4},
{-3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3, -1,  1, -4, -3, -2, 11,  2, -3, -4, -3, -2, -4},
{-2, -2, -2, -3, -2, -1, -2, -3,  2, -1, -1, -2, -1,  3, -3, -2, -2,  2,  7, -1, -3, -2, -1, -4},
{ 0, -3, -3, -3, -1, -2, -2, -3, -3,  3,  1, -2,  1, -1, -2, -2,  0, -3, -1,  4, -3, -2, -1, -4},
{-2, -1,  3,  4, -3,  0,  1, -1,  0, -3, -4,  0, -3, -3, -2,  0, -1, -4, -3, -3,  4,  1, -1, -4},
{-1,  0,  0,  1, -3,  3,  4, -2,  0, -3, -3,  1, -1, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
{ 0, -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2,  0,  0, -2, -1, -1, -1, -1, -1, -4},
{-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1}
};

double gettime() {
  struct timeval t;
  gettimeofday(&t,NULL);
  return t.tv_sec+t.tv_usec*1e-6;
}

long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{

	printf("WG size of kernel = %d \n", BLOCK_SIZE);

	runTest( argc, argv);

	return EXIT_SUCCESS;
}

void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <max_rows/max_cols> <penalty> \n", argv[0]);
	fprintf(stderr, "\t<dimension>  - x and y dimensions\n");
	fprintf(stderr, "\t<penalty> - penalty(positive integer)\n");
	exit(1);
}

void runTest( int argc, char** argv) 
{
	int max_rows, max_cols, penalty;
	int *input_itemsets, *output_itemsets, *referrence;
	int *matrix_cuda,  *referrence_cuda;
	int size;
	

	// the lengths of the two sequences should be able to divided by 16.
	// And at current stage  max_rows needs to equal max_cols
	if (argc == 3)
	{
		max_rows = atoi(argv[1]);
		max_cols = atoi(argv[1]);
		penalty = atoi(argv[2]);
	}
	else{
		usage(argc, argv);
	}
	
	if(atoi(argv[1])%16!=0){
		fprintf(stderr,"The dimension values must be a multiple of 16\n");
		exit(1);
	}
	

	max_rows = max_rows + 1;
	max_cols = max_cols + 1;
	referrence = (int *)malloc( max_rows * max_cols * sizeof(int) );
	input_itemsets = (int *)malloc( max_rows * max_cols * sizeof(int) );
	output_itemsets = (int *)malloc( max_rows * max_cols * sizeof(int) );
	

	if (!input_itemsets)
		fprintf(stderr, "error: can not allocate memory");

    srand ( 7 );
	
	
    for (int i = 0 ; i < max_cols; i++){
		for (int j = 0 ; j < max_rows; j++){
			input_itemsets[i*max_cols+j] = 0;
		}
	}
	
	printf("Start Needleman-Wunsch\n");
	
	for( int i=1; i< max_rows ; i++){    //please define your own sequence. 
       input_itemsets[i*max_cols] = rand() % 10 + 1;
	}
    for( int j=1; j< max_cols ; j++){    //please define your own sequence.
       input_itemsets[j] = rand() % 10 + 1;
	}


	for (int i = 1 ; i < max_cols; i++){
		for (int j = 1 ; j < max_rows; j++){
		referrence[i*max_cols+j] = blosum62[input_itemsets[i*max_cols]][input_itemsets[j]];
		}
	}

    for( int i = 1; i< max_rows ; i++)
       input_itemsets[i*max_cols] = -i * penalty;
	for( int j = 1; j< max_cols ; j++)
       input_itemsets[j] = -j * penalty;

long long time0;
long long time1;
long long time2;
long long time3;
long long time4;
long long time5;
long long time6;

time0 = get_time();
time1 = get_time();

	size = max_cols * max_rows;
	hipError_t err;
	err = hipMalloc((void**)& referrence_cuda, sizeof(int)*size);
	if (err != hipSuccess)
	{
		printf("Failed to allocate CUDA device buffer!\n");
		exit(-1);
	}
	err = hipMalloc((void**)& matrix_cuda, sizeof(int)*size);
	if (err != hipSuccess)
	{
		printf("Failed to allocate CUDA device buffer!\n");
		exit(-1);
	}
	
time2 = get_time();

	hipMemcpy(referrence_cuda, referrence, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(matrix_cuda, input_itemsets, sizeof(int) * size, hipMemcpyHostToDevice);

	dim3 dimGrid;
	dim3 dimBlock(BLOCK_SIZE, 1);
	int block_width = ( max_cols - 1 )/BLOCK_SIZE;

time3 = get_time();

	#pragma omp parallel num_threads(2) shared(flag)
	{
		if (omp_get_thread_num() == 0)
		{
			power = GetPowerGPU(&flag, 0);
		}
		else
		{
			#pragma omp barrier
#ifdef FOR
			for(int k = 0; k < 20; k++)
			{
				if (k == 0) // if for loop for power measurement is enabled, only first run is timed
				{
#endif
					GetTime(start);
#ifdef FOR
				}
#endif
				//process top-left matrix
				for( int i = 1 ; i <= block_width ; i++)
				{
					dimGrid.x = i;
					dimGrid.y = 1;
					needle_cuda_shared_1<<<dimGrid, dimBlock>>>(referrence_cuda, matrix_cuda
									,max_cols, penalty, i, block_width); 
				}
				hipDeviceSynchronize();

				//process bottom-right matrix
				for( int i = block_width - 1  ; i >= 1 ; i--)
				{
					dimGrid.x = i;
					dimGrid.y = 1;
					needle_cuda_shared_2<<<dimGrid, dimBlock>>>(referrence_cuda, matrix_cuda
									,max_cols, penalty, i, block_width); 
				}
				hipDeviceSynchronize();
#ifdef FOR
				if (k == 0)
				{
#endif
					GetTime(end);
#ifdef FOR
				}
			}
#endif
			flag = 1;
		}
	}
	totalTime = TimeDiff(start, end);
	energy = GetEnergyGPU(power, totalTime);

time4 = get_time();

    hipMemcpy(output_itemsets, matrix_cuda, sizeof(int) * size, hipMemcpyDeviceToHost);
	
time5 = get_time();

	hipFree(referrence_cuda);
	hipFree(matrix_cuda);

time6 = get_time();

printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",
	   (float) (time1-time0) / 1000000, (float) (time1-time0) / (float) (time6-time0) * 100);
printf("%15.12f s, %15.12f % : GPU MEM: ALO\n",
	   (float) (time2-time1) / 1000000, (float) (time2-time1) / (float) (time6-time0) * 100);
printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",
	   (float) (time3-time2) / 1000000, (float) (time3-time2) / (float) (time6-time0) * 100);

printf("%15.12f s, %15.12f % : GPU: KERNEL\n",
	   (float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time6-time0) * 100);

printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",
	   (float) (time5-time4) / 1000000, (float) (time5-time4) / (float) (time6-time0) * 100);
printf("%15.12f s, %15.12f % : GPU MEM: FRE\n",
	   (float) (time6-time5) / 1000000, (float) (time6-time5) / (float) (time6-time0) * 100);

printf("Total time: %.12f s\n", (float) (time6-time0) / 1000000);

    printf("\nComputation done in %0.3lf ms.\n", totalTime);
    if (power != -1) // -1 --> failed to read energy values
    {
        printf("Total energy used is %0.3lf jouls.\n", energy);
        printf("Average power consumption is %0.3lf watts.\n", power);
    }

//#define TRACEBACK
#ifdef TRACEBACK
	
	FILE *fpo = fopen("result.txt","w");
	fprintf(fpo, "print traceback value GPU:\n");
    
	for (int i = max_rows - 2,  j = max_rows - 2; i>=0, j>=0;){
		int nw, n, w, traceback;
		if ( i == max_rows - 2 && j == max_rows - 2 )
			fprintf(fpo, "%d ", output_itemsets[ i * max_cols + j]); //print the first element
		if ( i == 0 && j == 0 )
           break;
		if ( i > 0 && j > 0 ){
			nw = output_itemsets[(i - 1) * max_cols + j - 1];
		    w  = output_itemsets[ i * max_cols + j - 1 ];
            n  = output_itemsets[(i - 1) * max_cols + j];
		}
		else if ( i == 0 ){
		    nw = n = LIMIT;
		    w  = output_itemsets[ i * max_cols + j - 1 ];
		}
		else if ( j == 0 ){
		    nw = w = LIMIT;
            n  = output_itemsets[(i - 1) * max_cols + j];
		}
		else{
		}

		//traceback = maximum(nw, w, n);
		int new_nw, new_w, new_n;
		new_nw = nw + referrence[i * max_cols + j];
		new_w = w - penalty;
		new_n = n - penalty;
		
		traceback = maximum(new_nw, new_w, new_n);
		if(traceback == new_nw)
			traceback = nw;
		if(traceback == new_w)
			traceback = w;
		if(traceback == new_n)
            traceback = n;
			
		fprintf(fpo, "%d ", traceback);

		if(traceback == nw )
		{i--; j--; continue;}

        else if(traceback == w )
		{j--; continue;}

        else if(traceback == n )
		{i--; continue;}

		else
		;
	}
	
	fclose(fpo);

#endif

	free(referrence);
	free(input_itemsets);
	free(output_itemsets);
	
}

