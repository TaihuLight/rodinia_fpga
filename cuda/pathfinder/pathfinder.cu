#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#define BLOCK_SIZE 256
#define STR_SIZE 256
#define DEVICE 0
#define HALO 1 // halo width along one direction when advancing to the next iteration

void run(int argc, char** argv);

int rows, cols;
int* data;
int** wall;
int* result;
#define M_SEED 9
int pyramid_height;
FILE *resultFile;

#include <omp.h>
#include "../../common/timer.h"
#include "../../common/power_gpu.h"

TimeStamp start, end;
double totalTime;
int flag = 0;
double power = 0;
double energy = 0;

//#define BENCH_PRINT

void
init(int argc, char** argv)
{
	if(argc==4){

		cols = atoi(argv[1]);

		rows = atoi(argv[2]);

                pyramid_height=atoi(argv[3]);
	}else{
                printf("Usage: dynproc row_len col_len pyramid_height\n");
                exit(0);
        }
	data = new int[rows*cols];

	wall = new int*[rows];

	for(int n=0; n<rows; n++)

		wall[n]=data+cols*n;

	result = new int[cols];

	

	int seed = M_SEED;

	srand(seed);



	for (int i = 0; i < rows; i++)

    {

        for (int j = 0; j < cols; j++)

        {

            wall[i][j] = rand() % 10;

        }

    }

#ifdef BENCH_PRINT

    for (int i = 0; i < rows; i++)

    {

        for (int j = 0; j < cols; j++)

        {

            fprintf(resultFile, "%d ",wall[i][j]) ;

        }

        fprintf(resultFile, "\n") ;

    }

#endif
}

void 
fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void dynproc_kernel(
                int iteration, 
                int *gpuWall,
                int *gpuSrc,
                int *gpuResults,
                int cols, 
                int rows,
                int startStep,
                int border)
{

        __shared__ int prev[BLOCK_SIZE];
        __shared__ int result[BLOCK_SIZE];

	int bx = blockIdx.x;
	int tx=threadIdx.x;
	
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data

        // calculate the small block size
	int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkX = small_block_cols*bx-border;
        int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int xidx = blkX+tx;
       
        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

        int W = tx-1;
        int E = tx+1;
        
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;

        bool isValid = IN_RANGE(tx, validXmin, validXmax);

	if(IN_RANGE(xidx, 0, cols-1)){
            prev[tx] = gpuSrc[xidx];
	}
	__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
        bool computed;
        for (int i=0; i<iteration ; i++){ 
            computed = false;
            if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                  isValid){
                  computed = true;
                  int left = prev[W];
                  int up = prev[tx];
                  int right = prev[E];
                  int shortest = MIN(left, up);
                  shortest = MIN(shortest, right);
                  int index = cols*(startStep+i)+xidx;
                  result[tx] = shortest + gpuWall[index];
	
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)	 //Assign the computation range
                prev[tx]= result[tx];
	    __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
      }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
          gpuResults[xidx]=result[tx];		
      }
}

/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols, \
	 int pyramid_height, int blockCols, int borderCols)
{
        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid(blockCols);
	
        int src = 1, dst = 0;
	
	hipDeviceSynchronize();
	
	#pragma omp parallel num_threads(2) shared(flag)
	{
		if (omp_get_thread_num() == 0)
		{
			power = GetPowerGPU(&flag, 0);
		}
		else
		{
			#pragma omp barrier
#ifdef FOR
			for(int k = 0; k < 10000; k++)
			{
				if (k == 0)
				{
#endif
					GetTime(start);
#ifdef FOR
				}
#endif
				for (int t = 0; t < rows-1; t+=pyramid_height)
				{
					int temp = src;
					src = dst;
					dst = temp;
					dynproc_kernel<<<dimGrid, dimBlock>>>(
						MIN(pyramid_height, rows-t-1), 
						gpuWall, gpuResult[src], gpuResult[dst],
						cols,rows, t, borderCols);
				}
				hipDeviceSynchronize();
#ifdef FOR
				if (k == 0)
				{
#endif
					GetTime(end);
#ifdef FOR
				}
			}
#endif
			flag = 1;
		}
	}
	totalTime = TimeDiff(start, end);
	energy = GetEnergyGPU(power, totalTime);

        return dst;
}

int main(int argc, char** argv)
{
    int num_devices;
    hipGetDeviceCount(&num_devices);
    if (num_devices > 1) hipSetDevice(DEVICE);
    
    resultFile = fopen("result.txt", "w");
    if (resultFile == NULL)
    {
        printf("Failed to open result file!\n");
        exit(-1);
    }

    run(argc,argv);
    
    printf("\nComputation done in %0.3lf ms.\n", totalTime);
    if (power != -1) // -1 --> failed to read energy values
    {
        printf("Total energy used is %0.3lf jouls.\n", energy);
        printf("Average power consumption is %0.3lf watts.\n", power);
    }

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    init(argc, argv);

    /* --------------- pyramid parameters --------------- */
    int borderCols = (pyramid_height)*HALO;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*HALO*2;
    int blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);

    fprintf(resultFile, "pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",\
	pyramid_height, cols, borderCols, BLOCK_SIZE, blockCols, smallBlockCol);
	
    int *gpuWall, *gpuResult[2];
    int size = rows*cols;

    hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
    hipMemcpy(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice);
    hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
    hipMemcpy(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice);


    int final_ret = calc_path(gpuWall, gpuResult, rows, cols, \
	 pyramid_height, blockCols, borderCols);

    hipMemcpy(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost);


#ifdef BENCH_PRINT

    for (int i = 0; i < cols; i++)

            fprintf(resultFile, "%d ",data[i]) ;

    fprintf(resultFile, "\n") ;

#endif
    
    for (int i = 0; i < cols; i++)

            fprintf(resultFile, "%d ",result[i]) ;

    fprintf(resultFile, "\n") ;


    hipFree(gpuWall);
    hipFree(gpuResult[0]);
    hipFree(gpuResult[1]);

    delete [] data;
    delete [] wall;
    delete [] result;
    fclose(resultFile);

}

